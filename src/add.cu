#include "hip/hip_runtime.h"
#include "add.h"
#include "kernel.h"
#include "device_util.h"

#include <stdio.h>

namespace {
    __global__ void add(int a, int b, int *c) {
        *c = a + b;
    }
}

namespace cufoo {
namespace kernels
{
    template <> status add::run<compute_mode::CUDA>(
        int a, int b, int* c
        )
    {
        int *dev_c;
        checkCudaErrors(hipMalloc((void**)&dev_c, sizeof(int)));

        ::add<<<1,1>>>(a, b, dev_c);
        if (hipPeekAtLastError() != hipSuccess) {
            printf("[E] %s", hipGetErrorString(hipGetLastError()));
            return status::KERNEL_FAILED;
        }

        hipDeviceSynchronize();
        checkCudaErrors(hipMemcpy(c, dev_c, sizeof(int), hipMemcpyDeviceToHost));
        checkCudaErrors(hipFree(dev_c));

        return status::SUCCESS;
    }
}
}