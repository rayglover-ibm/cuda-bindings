#include "hip/hip_runtime.h"
#include "add.h"
#include "kernel.h"
#include "device_util.h"

#include <gsl.h>
#include <stdio.h>

namespace
{
    __global__ void add(int a, int b, gsl::span<int> result) {
        result[0] = a + b;
    }

    __global__ void add_span(
        gsl::span<int> a, gsl::span<int> b, gsl::span<int> result
        )
    {
        int idx = threadIdx.x + blockIdx.x * blockDim.x;

        if (idx < result.size()) {
            result[idx] = a[idx] + b[idx];
        }
    }
}

namespace cufoo {
namespace kernels
{
    using device_util::dev_ptr;

    template <> status add::run<compute_mode::CUDA>(
        int a, int b, int* c
        )
    {
        dev_ptr<int> dev_c(1);

        ::add<<< 1, 1 >>>(a, b, dev_c.span());
        if (!checkCudaLastError()) return status::KERNEL_FAILED;

        hipDeviceSynchronize();
        dev_c.copy_to({ c, 1 });

        return status::SUCCESS;
    }

    template <> status add::run<compute_mode::CUDA>(
        gsl::span<int> a, gsl::span<int> b, gsl::span<int> result
        )
    {
        size_t N = result.size();

        dev_ptr<int> dev_a(a);
        dev_ptr<int> dev_b(b);
        dev_ptr<int> dev_result(N);

        int blockSize, minGridSize, gridSize;

        hipOccupancyMaxPotentialBlockSize(
            &minGridSize, &blockSize, ::add_span, 0, 0);

        /* Round up according to array size */
        gridSize = (N + blockSize - 1) / blockSize;

        ::add_span<<< gridSize, blockSize >>>(
            dev_a.span(), dev_b.span(), dev_result.span());

        if (!checkCudaLastError()) return status::KERNEL_FAILED;

        hipDeviceSynchronize();
        dev_result.copy_to(result);

        return status::SUCCESS;
    }
}
}